#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <float.h>

#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/sampling_layers/TopKSamplingLayer.h"
#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"

namespace fastertransformer {

    #include <unistd.h>
#include <sys/syscall.h>

#define __FILENAME__ (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

template<uint TOP_K_MAX>
__global__ void setup_topk_runtime_args(int    batch_size,
                                        uint   top_k,
                                        uint*  top_ks,
                                        int    top_ks_size,
                                        float  top_p,
                                        float* top_ps,
                                        int    top_ps_size,
                                        bool*  skip_decode)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x) {
        uint  k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f) {
            // FT's topp implementation does not support topp = 0.0f, but it equivalent to greedy search.
            // So, we set the topk = 1 as an alternative solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f) {
            // for compatibility <= FT5.0.
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX=64.
        top_ks[i] = k > TOP_K_MAX ? TOP_K_MAX : k;
        if (k > TOP_K_MAX) {
            printf("[WARNING] topk (%d) is larger than max supported number (%d) for token %d"
                   " clip to max supported number %d. \n",
                   k,
                   TOP_K_MAX,
                   i,
                   top_ks[i]);
        }
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f) {
            printf("[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                   " clip to closest number %f.\n",
                   p,
                   i,
                   top_ps[i]);
        }
        skip_decode[i] = k == 0;
    }
}

template<typename T>
void TopKSamplingLayer<T>::allocateBuffer()
{
    FT_CHECK(false);
}

template<typename T>
void TopKSamplingLayer<T>::allocateBuffer(size_t batch_size, Tensor top_k, Tensor top_p)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::allocateBuffer(batch_size, top_k, top_p);
    uint max_top_k = top_k.size() > 0 ? top_k.max<uint>() : 1;
    if (max_top_k == 0) {
        // for safety. TopKSamplingLayer handles a case of top_k=0 and top_p=0 as
        // a greedy decode, i.e. top_k=1, although such case has max_top_k=0.
        max_top_k = 1;
    }
    printf("--------allocateBuffer: 1 sampling_workspace_size_=%d\n",sampling_workspace_size_);
    invokeTopKSampling<T>(nullptr,
                          sampling_workspace_size_,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          max_top_k,
                          1.0f,
                          vocab_size_padded_,
                          nullptr,
                          stream_,
                          batch_size,
                          skip_decode_buf_);
    printf("--------allocateBuffer: 2 sampling_workspace_size_=%d\n",sampling_workspace_size_);
    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, false);
    runtime_top_k_buf_ =
        reinterpret_cast<uint*>(allocator_->reMalloc(runtime_top_k_buf_, sizeof(uint) * batch_size, false));
    runtime_top_p_buf_ =
        reinterpret_cast<float*>(allocator_->reMalloc(runtime_top_p_buf_, sizeof(float) * batch_size, false));
    is_allocate_buffer_ = true;
}

template<typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_) {
        allocator_->free((void**)(&sampling_workspace_));
        allocator_->free((void**)(&runtime_top_k_buf_));
        allocator_->free((void**)(&runtime_top_p_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    is_allocate_buffer_ = false;
}

template<typename T>
void TopKSamplingLayer<T>::setup(const size_t batch_size, const size_t beam_width, TensorMap* runtime_args)
{
    // Setup runtime topk and topp arguments.
    //
    // runtime_args:
    //     runtime_top_k [1] or [batch_size] on cpu, optional, uint.
    //     runtime_top_p [1] or [batch_size] on cpu, optional, float.
    //     temperature [1] or [batch_size] on cpu, optional
    //     repetition_penalty [1] or [batch_size] on cpu, optional
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setup(batch_size, beam_width, runtime_args);

    uint         tmp_top_k     = 0;
    const Tensor runtime_top_k = runtime_args->isExist("runtime_top_k") ?
                                     runtime_args->at("runtime_top_k") :
                                     Tensor(MEMORY_CPU, TYPE_UINT32, {1}, &tmp_top_k);
    const Tensor runtime_top_p = runtime_args->isExist("runtime_top_p") ? runtime_args->at("runtime_top_p") : Tensor();
    const size_t runtime_top_k_size = runtime_top_k.size();
    const size_t runtime_top_p_size = runtime_top_p.size();

    uint  top_k = runtime_top_k.max<uint>();
    float top_p = runtime_top_p_size == 0 ? 0.0f : runtime_top_p.getVal<float>();

    if (runtime_top_k_size > 1) {
        FT_CHECK_WITH_INFO(
            runtime_top_k.size() == batch_size,
            fmtstr("runtime_top_k.size() (%d) == batch_size (%d) is not satisfied!", runtime_top_k.size(), batch_size));
        cudaAutoCpy(runtime_top_k_buf_, runtime_top_k.getPtr<uint>(), batch_size, stream_);
    }
    if (runtime_top_p_size > 1) {
        FT_CHECK_WITH_INFO(
            runtime_top_p.size() == batch_size,
            fmtstr("runtime_top_p.size() (%d) == batch_size (%d) is not satisfied!", runtime_top_p.size(), batch_size));
        cudaAutoCpy(runtime_top_p_buf_, runtime_top_p.getPtr<float>(), batch_size, stream_);
    }

    dim3 block(std::min((int)batch_size, 256));
    dim3 grid(div_up((int)batch_size, (int)block.x));
    // support top_k up to 1024.
    setup_topk_runtime_args<1024><<<grid, block, 0, stream_>>>(batch_size,
                                                               top_k,
                                                               runtime_top_k_buf_,
                                                               runtime_top_k_size,
                                                               top_p,
                                                               runtime_top_p_buf_,
                                                               runtime_top_p_size,
                                                               skip_decode_buf_);
    cudaAutoCpy(skip_decode_, skip_decode_buf_, batch_size, stream_);
    uint* runtime_top_ks = new uint[batch_size];
    cudaAutoCpy(runtime_top_ks, runtime_top_k_buf_, batch_size, stream_);
    runtime_max_top_k_ = static_cast<int>(*std::max_element(runtime_top_ks, runtime_top_ks + batch_size));
    delete[] runtime_top_ks;
}

#include <unistd.h>
#include <sys/syscall.h>

#define __FILENAME__ (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)


template<typename T>
void TopKSamplingLayer<T>::runSampling(TensorMap* output_tensors, TensorMap* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, vocab_size_padded]
    //      embedding_bias [vocab_size_padded], optional
    //      step [1] on cpu
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size], optional
    //      ite [1] on cpu

    // output_tensors:
    //      output_ids [max_seq_len, batch_size]
    //      finished [local_batch_size], optional
    //      sequence_length [local_batch_size], optional
    //      cum_log_probs [batch_size], must be float*, optional
    //          The cumultative log probability of generated tokens.
    //      output_log_probs [local_batch_size], must be float*, optional
    //          The log probs at the current step.

    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    FT_CHECK(input_tensors->size() >= 4);
    FT_CHECK(output_tensors->size() >= 1);

    const int batch_size       = output_tensors->at("output_ids").shape[1];
    const int local_batch_size = input_tensors->at("logits").shape[0];
    const int ite              = input_tensors->at("ite").getVal<int>();
    const int step             = input_tensors->at("step").getVal<int>();

    // in case of skip any, the logit value is already copied and processed.
    T* logits = !skip_any_ ? input_tensors->at("logits").getPtr<T>() : runtime_logits_buf_;

    printf("[%s:%d-pid:%d-%s]: <......invokeAddBiasEndMask.....> local_batch_size=%d vocab_size_=%d vocab_size_padded_=%d\n", 
        __FUNCTION__, __LINE__, getpid(),__FILENAME__,
        local_batch_size,
        vocab_size_,
        vocab_size_padded_);
    //赋值结束标志
    invokeAddBiasEndMask(logits,//out
                         (T*)(nullptr),//in bias
                         input_tensors->at("end_id").getPtr<const int>(),//in end_ids
                         output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),//in finished
                         local_batch_size,
                         vocab_size_,
                         vocab_size_padded_,
                         stream_);
    sync_check_cuda_error();

    float* cum_log_probs =
        output_tensors->isExist("cum_log_probs") ? output_tensors->at("cum_log_probs").getPtr<float>() : nullptr;
    float* output_log_probs =
        output_tensors->isExist("output_log_probs") ? output_tensors->at("output_log_probs").getPtr<float>() : nullptr;

    if (cum_log_probs != nullptr || output_log_probs != nullptr) {
        printf("[%s:%d-pid:%d-%s]: <......invokeAddBiasSoftMax.....>\n", 
            __FUNCTION__, __LINE__, getpid(),__FILENAME__);
        invokeAddBiasSoftMax(
            logits,
            (T*)(nullptr),
            input_tensors->at("end_id").getPtr<int>(),
            output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
            local_batch_size,
            vocab_size_padded_,
            vocab_size_,
            stream_);
        sync_check_cuda_error();
    }

    printf("\n");
    printf("\n");
    printf("step:%d\n",step);
    {
        int out_size = output_tensors->at("output_ids").size();
        int offset = step * batch_size + ite * local_batch_size;
        printf("B-----------------output_ids[%d/%d] [runSampling] step=%d batch:%d ite:%d  local_batch:%d\n", 
            out_size,offset,step,batch_size,ite,local_batch_size);

        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("output_ids").getPtrWithOffset<int>(0), out_size);
        printf("B-----------------output_ids[size:%d,off:%d] [runSampling] h_buf:\n", out_size,offset);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = output_tensors->at("sequence_length").size();
        printf("B-----------------sequence_length[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("sequence_length").getPtr<int>(), out_size);
        printf("B-----------------sequence_length[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = output_tensors->at("finished").size();
        printf("B-----------------finished[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("finished").getPtr<int>(), out_size);
        printf("B-----------------finished[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = input_tensors->at("end_id").size();
        printf("B-----------------end_id[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), input_tensors->at("end_id").getPtr<int>(), out_size);
        printf("B-----------------end_id[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    if (output_log_probs != NULL)
    {
        int out_size = output_tensors->at("output_log_probs").size();
        printf("B-----------------output_log_probs[%d] [runSampling]\n", out_size);
        std::vector<float> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("output_log_probs").getPtr<float>(), out_size);
        printf("B-----------------output_log_probs[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%f ", h_buf[k]);
        } 
        printf("\n");
    }

    if (cum_log_probs != NULL)
    {
        int out_size = output_tensors->at("cum_log_probs").size();
        printf("B-----------------cum_log_probs[%d] [runSampling]\n", out_size);
        std::vector<float> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("cum_log_probs").getPtr<float>(), out_size);
        printf("B-----------------cum_log_probs[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%f ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = input_tensors->at("logits").size();
        printf("B-----------------logits[%d] [runSampling]\n", out_size);
        std::vector<T> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), input_tensors->at("logits").getPtr<T>(), out_size);
        

        float sum_log = 0.0;
        for(int k=0; k<out_size; k++) {
            sum_log += (float)h_buf[k];
        }

        printf("B-----------------logits[%d] [runSampling] sum_log[%f] h_buf:\n", sum_log, out_size);

        for(int k=0; k<(out_size>1024?100:out_size); k++) {
            printf("%.2f ", h_buf[k]);
        } 
        printf("\n");
    }


    printf("[%s:%d-pid:%d-%s]: <......invokeBatchTopKSampling.....> sampling_workspace_size_=%d curandstate_buf_=%p\n", 
        __FUNCTION__, __LINE__, getpid(),__FILENAME__,sampling_workspace_size_,curandstate_buf_);
    invokeBatchTopKSampling(
        sampling_workspace_,
        sampling_workspace_size_,//201280=sizeof(T)[4]*temp_log_probs_buf_size[50304]+sizeof(int)[4]*topk_tmp_ids_buf_size[8]+ sizeof(T)[4]*topk_tmp_val_buf_size[8]
        //
        logits,//input_tensors->at("logits").getPtr<T>()
        output_tensors->at("output_ids").getPtrWithOffset<int>(step * batch_size + ite * local_batch_size),//output_ids 存储输出token
        output_tensors->at("sequence_length", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<int>(),//sequence_length
        output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),//finished
        cum_log_probs, // NULL
        output_log_probs,//NULL
        curandstate_buf_ + ite * local_batch_size,
        (int)runtime_max_top_k_,  // useless because runtime_top_k_buf_ is never nullptr. Keep for legacy.
        (int*)(runtime_top_k_buf_ + ite * local_batch_size),//[batch]
        1.0f,  // useless because runtime_top_p_buf_ is never nullptr. Keep for legacy.
        runtime_top_p_buf_ + ite * local_batch_size, //[batch] float
        vocab_size_padded_,//50304
        input_tensors->at("end_id").getPtr<int>(),//end_id
        stream_,
        local_batch_size,//1
        skip_decode_buf_ + ite * local_batch_size);//[batch]
    sync_check_cuda_error();

    {
        int out_size = input_tensors->at("logits").size();
        printf("E-----------------logits[%d] [runSampling]\n", out_size);
        std::vector<T> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), input_tensors->at("logits").getPtr<T>(), out_size);
        printf("E-----------------logits[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<(out_size>1024?100:out_size); k++) {
            printf("%.2f ", h_buf[k]);
        } 
        printf("\n");
    }

    if (cum_log_probs != NULL)
    {
        int out_size = output_tensors->at("cum_log_probs").size();
        printf("E-----------------cum_log_probs[%d] [runSampling]\n", out_size);
        std::vector<float> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("cum_log_probs").getPtr<float>(), out_size);
        printf("E-----------------cum_log_probs[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%f ", h_buf[k]);
        } 
        printf("\n");
    }

    if (output_log_probs != NULL)
    {
        int out_size = output_tensors->at("output_log_probs").size();
        printf("E-----------------output_log_probs[%d] [runSampling]\n", out_size);
        std::vector<float> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("output_log_probs").getPtr<float>(), out_size);
        printf("E-----------------output_log_probs[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%f ", h_buf[k]);
        } 
        printf("\n");
    }

     {
        int out_size = input_tensors->at("end_id").size();
        printf("E-----------------end_id[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), input_tensors->at("end_id").getPtr<int>(), out_size);
        printf("E-----------------end_id[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = output_tensors->at("finished").size();
        printf("E-----------------finished[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("finished").getPtr<int>(), out_size);
        printf("E-----------------finished[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = output_tensors->at("sequence_length").size();
        printf("E-----------------sequence_length[%d] [runSampling]\n", out_size);
        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("sequence_length").getPtr<int>(), out_size);
        printf("E-----------------sequence_length[%d] [runSampling] h_buf:\n", out_size);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }

    {
        int out_size = output_tensors->at("output_ids").size();
        int offset = step * batch_size + ite * local_batch_size;
        printf("E-----------------output_ids[%d/%d] [runSampling] step=%d batch:%d ite:%d  local_batch:%d\n", 
            out_size,offset,step,batch_size,ite,local_batch_size);

        std::vector<int> h_buf(out_size);
        cudaD2Hcpy(h_buf.data(), output_tensors->at("output_ids").getPtrWithOffset<int>(0), out_size);
        printf("E-----------------output_ids[size:%d,off:%d] [runSampling] h_buf:\n", out_size,offset);
        for(int k=0; k<out_size; k++) {
            printf("%d ", h_buf[k]);
        } 
        printf("\n");
    }
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t             max_batch_size,
                                        size_t             vocab_size,
                                        size_t             vocab_size_padded,
                                        int                end_id,
                                        size_t             top_k,
                                        unsigned long long random_seed,
                                        float              temperature,
                                        float              len_penalty,
                                        float              repetition_penalty,
                                        hipStream_t       stream,
                                        cublasMMWrapper*   cublas_wrapper,
                                        IAllocator*        allocator,
                                        bool               is_free_buffer_after_forward):
    BaseSamplingLayer<T>(max_batch_size,
                         vocab_size,
                         vocab_size_padded,
                         end_id,
                         top_k,
                         0.0f,
                         random_seed,
                         temperature,
                         len_penalty,
                         repetition_penalty,
                         stream,
                         cublas_wrapper,
                         allocator,
                         is_free_buffer_after_forward,
                         nullptr)
{
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(TopKSamplingLayer<T> const& top_k_sampling_layer):
    BaseSamplingLayer<T>(top_k_sampling_layer)
{
    printf("--------TopKSamplingLayer: sampling_workspace_size_=%d\n",sampling_workspace_size_);
}

template<typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

}  // namespace fastertransformer
